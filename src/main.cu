
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void main_cuda(float *d_v1, float *d_v2, float *d_result) {
    int i = threadIdx.x;
    d_result[i] = d_v1[i] + d_v2[i];
}

int main(int argc, char *argv[]) {
    printf("Hello, World!\n");

    printf("Arg0: %s\n", argv[0]);
    printf("Arg1: %s\n", argv[1]);
    printf("Arg2: %s\n", argv[2]);

    int n = 1024;
    int n_bytes = sizeof(float) * n;

    dim3 grid = dim3(1, 1, 1);
    dim3 block = dim3(n, 1, 1);

    float *h_v1 = (float *) malloc(n_bytes);
    float *h_v2 = (float *) malloc(n_bytes);
    float *h_result = (float *) malloc(n_bytes);

    float *d_v1;
    float *d_v2;
    float *d_result;

    for (int i = 0; i < n; ++i) {
        h_v1[i] = i;
        h_v2[i] = i;
    }

    hipMalloc((void **) &d_v1, n_bytes);
    hipMalloc((void **) &d_v2, n_bytes);
    hipMalloc((void **) &d_result, n_bytes);

    hipMemcpy(d_v1, h_v1, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_v2, h_v2, n_bytes, hipMemcpyHostToDevice);

    main_cuda<<<grid,block>>>(d_v1, d_v2, d_result);

    hipEvent_t event;

    hipEventCreate(&event);
    hipEventRecord(event, 0);
    hipEventSynchronize(event);

    hipMemcpy(h_result, d_result, n_bytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i) {
        printf("%4d: %4.1f\n", i, h_result[i]);
    }

    hipEventDestroy(event);

    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_result);

    free(h_v1);
    free(h_v2);
    free(h_result);

    return 0;
}